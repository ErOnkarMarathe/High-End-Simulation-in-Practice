#include <iostream>
#include <fstream>
#include <cmath>
#include <iomanip>
#include <random>
#include <string>
#include <hip/hip_runtime.h>

// Custom atomicAdd for double precision
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// Helper function to calculate the distance between two particles in 3D
__device__ double distance(const double* pos1, const double* pos2, const double* domain) {
    double dx = pos2[0] - pos1[0];
    double dy = pos2[1] - pos1[1];
    double dz = pos2[2] - pos1[2];

    // Apply periodic boundary conditions
    if (dx > domain[0] / 2.0) dx -= domain[0];
    else if (dx < -domain[0] / 2.0) dx += domain[0];
    
    if (dy > domain[1] / 2.0) dy -= domain[1];
    else if (dy < -domain[1] / 2.0) dy += domain[1];
    
    if (dz > domain[2] / 2.0) dz -= domain[2];
    else if (dz < -domain[2] / 2.0) dz += domain[2];

    // Compute the minimum distance using the adjusted coordinates
    return sqrt(dx * dx + dy * dy + dz * dz);
}

__global__ void calculateForces(int numParticles, double* positions, double* forces, double sigma, double epsilon, double* domain, double cutoff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        double force[3] = {0.0, 0.0, 0.0};
        for (int j = 0; j < numParticles; ++j) {
            if (i != j) {
                double r = distance(&positions[i * 3], &positions[j * 3], domain); // Pass domain to distance function
                if (r != 0 && r < cutoff) {
                    double coeff = 24 * epsilon / r * (2 * pow(sigma / r, 12) - pow(sigma / r, 6));
                    double dx = positions[i * 3 + 0] - positions[j * 3 + 0];
                    double dy = positions[i * 3 + 1] - positions[j * 3 + 1];
                    double dz = positions[i * 3 + 2] - positions[j * 3 + 2];

                    // Apply minimum image convention
                    dx -= domain[0] * round(dx / domain[0]);
                    dy -= domain[1] * round(dy / domain[1]);
                    dz -= domain[2] * round(dz / domain[2]);

                    force[0] += coeff * dx / r;
                    force[1] += coeff * dy / r;
                    force[2] += coeff * dz / r;
                }
            }
        }
        forces[i * 3 + 0] = force[0];
        forces[i * 3 + 1] = force[1];
        forces[i * 3 + 2] = force[2];
    }
}

__global__ void updateParticles(int numParticles, double* positions, double* velocities, double* forces, double* masses, double dt, double* domain) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        double acc[3];
        acc[0] = forces[i * 3 + 0] / masses[i];
        acc[1] = forces[i * 3 + 1] / masses[i];
        acc[2] = forces[i * 3 + 2] / masses[i];

        positions[i * 3 + 0] += velocities[i * 3 + 0] * dt + 0.5 * acc[0] * dt * dt;
        positions[i * 3 + 1] += velocities[i * 3 + 1] * dt + 0.5 * acc[1] * dt * dt;
        positions[i * 3 + 2] += velocities[i * 3 + 2] * dt + 0.5 * acc[2] * dt * dt;

        velocities[i * 3 + 0] += 0.5 * acc[0] * dt;
        velocities[i * 3 + 1] += 0.5 * acc[1] * dt;
        velocities[i * 3 + 2] += 0.5 * acc[2] * dt;

        // Apply periodic boundary conditions
        positions[i * 3 + 0] = fmod(positions[i * 3 + 0] + domain[0], domain[0]);
        positions[i * 3 + 1] = fmod(positions[i * 3 + 1] + domain[1], domain[1]);
        positions[i * 3 + 2] = fmod(positions[i * 3 + 2] + domain[2], domain[2]);

        if (positions[i * 3 + 0] < 0) positions[i * 3 + 0] += domain[0];
        if (positions[i * 3 + 1] < 0) positions[i * 3 + 1] += domain[1];
        if (positions[i * 3 + 2] < 0) positions[i * 3 + 2] += domain[2];
    }
}

__global__ void finalizeVelocities(int numParticles, double* velocities, double* forces, double* masses, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        double acc[3];
        acc[0] = forces[i * 3 + 0] / masses[i];
        acc[1] = forces[i * 3 + 1] / masses[i];
        acc[2] = forces[i * 3 + 2] / masses[i];

        velocities[i * 3 + 0] += 0.5 * acc[0] * dt;
        velocities[i * 3 + 1] += 0.5 * acc[1] * dt;
        velocities[i * 3 + 2] += 0.5 * acc[2] * dt;
    }
}

__global__ void calculateTotalEnergy(int numParticles, double* positions, double* velocities, double* masses, double sigma, double epsilon, double* totalEnergy, double* domain, double cutoff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ double sharedEnergy[256]; // Assuming blockSize is 256
    sharedEnergy[threadIdx.x] = 0.0;

    // Calculate kinetic energy
    if (i < numParticles) {
        double kineticEnergy = 0.5 * masses[i] * (pow(velocities[i * 3 + 0], 2) + pow(velocities[i * 3 + 1], 2) + pow(velocities[i * 3 + 2], 2));
        sharedEnergy[threadIdx.x] += kineticEnergy;
    }

    __syncthreads();

    // Calculate potential energy
    if (i < numParticles) {
        for (int j = i + 1; j < numParticles; ++j) {
            double r = distance(&positions[i * 3], &positions[j * 3],domain);
            if (r <= cutoff) {
                double potentialEnergy = 4 * epsilon * (pow(sigma / r, 12) - pow(sigma / r, 6));
                sharedEnergy[threadIdx.x] += potentialEnergy;
            }
        }
    }

    __syncthreads();

    // Reduce shared memory to get total energy for this block
    if (threadIdx.x == 0) {
        double blockEnergy = 0.0;
        for (int k = 0; k < blockDim.x; ++k) {
            blockEnergy += sharedEnergy[k];
        }
        atomicAddDouble(totalEnergy, blockEnergy);
    }
}

void initializeParticles(int numParticles, double* positions, double* velocities, double* masses, int gridSize, double spacing, double mass, double* domain) {
    //std::random_device rd;
    std::mt19937 gen(42);
    std::normal_distribution<> d(0, 1);

    int count = 0;
    double offset = spacing / 2.0;
    for (int x = 0; x < gridSize; ++x) {
        for (int y = 0; y < gridSize; ++y) {
            for (int z = 0; z < gridSize; ++z) {
                if (count < numParticles) {
                    positions[count * 3 + 0] = x * spacing + offset;
                    positions[count * 3 + 1] = y * spacing + offset;
                    positions[count * 3 + 2] = z * spacing + offset;

                    // Ensure particles are within the domain boundaries
                    positions[count * 3 + 0] = fmod(positions[count * 3 + 0], domain[0] - 2 * offset) + offset;
                    positions[count * 3 + 1] = fmod(positions[count * 3 + 1], domain[1] - 2 * offset) + offset;
                    positions[count * 3 + 2] = fmod(positions[count * 3 + 2], domain[2] - 2 * offset) + offset;

                    velocities[count * 3 + 0] = d(gen);
                    velocities[count * 3 + 1] = d(gen);
                    velocities[count * 3 + 2] = d(gen);

                    masses[count] = mass;
                    ++count;
                }
            }
        }
    }
}

void writeBoxVertices(std::ofstream &vtkFile, const double* domain) {
    vtkFile << std::fixed << std::setprecision(6);
    vtkFile << 0.0 << " " << 0.0 << " " << 0.0 << "\n";
    vtkFile << domain[0] << " " << 0.0 << " " << 0.0 << "\n";
    vtkFile << domain[0] << " " << domain[1] << " " << 0.0 << "\n";
    vtkFile << 0.0 << " " << domain[1] << " " << 0.0 << "\n";
    vtkFile << 0.0 << " " << 0.0 << " " << domain[2] << "\n";
    vtkFile << domain[0] << " " << 0.0 << " " << domain[2] << "\n";
    vtkFile << domain[0] << " " << domain[1] << " " << domain[2] << "\n";
    vtkFile << 0.0 << " " << domain[1] << " " << domain[2] << "\n";
}

void writeBoxEdges(std::ofstream &vtkFile) {
    vtkFile << "LINES 12 36\n";
    vtkFile << "2 0 1\n2 1 2\n2 2 3\n2 3 0\n";
    vtkFile << "2 4 5\n2 5 6\n2 6 7\n2 7 4\n";
    vtkFile << "2 0 4\n2 1 5\n2 2 6\n2 3 7\n";
}


int main() {
    int numParticles;
    double sigma, epsilon, dt, time, mass,cutoff;
    double domain[3];

    std::cout << "Enter the number of particles: ";
    std::cin >> numParticles;

    // Allocate unified memory
    double* d_positions;
    double* d_velocities;
    double* d_forces;
    double* d_masses;
    double* d_totalEnergy;
    double* d_domain;

    hipMallocManaged(&d_positions, numParticles * 3 * sizeof(double));
    hipMallocManaged(&d_velocities, numParticles * 3 * sizeof(double));
    hipMallocManaged(&d_forces, numParticles * 3 * sizeof(double));
    hipMallocManaged(&d_masses, numParticles * sizeof(double));
    hipMallocManaged(&d_totalEnergy, sizeof(double));
    hipMallocManaged(&d_domain, 3 * sizeof(double));

    int gridSize = std::ceil(std::pow(numParticles, 1.0 / 3.0));
    double spacing = 1.0;

    std::cout << "Enter the mass of each particle: ";
    std::cin >> mass;

    std::cout << "Enter domain dimensions (x y z): ";
    std::cin >> domain[0] >> domain[1] >> domain[2];

    hipMemcpy(d_domain, domain, 3 * sizeof(double), hipMemcpyHostToDevice);

    initializeParticles(numParticles, d_positions, d_velocities, d_masses, gridSize, spacing, mass, domain);

    std::cout << "Enter Lennard-Jones constant sigma: ";
    std::cin >> sigma;
    std::cout << "Enter Lennard-Jones constant epsilon: ";
    std::cin >> epsilon;
    std::cout << "Enter size of time steps: ";
    std::cin >> dt;
    std::cout << "Enter the end time for simulation: ";
    std::cin >> time;
    std::cout << "Enter the cutoff distance: ";
    std::cin >> cutoff;

    int num_steps = time / dt;
    
    // Open initial VTK file
    std::ofstream vtkFile("particles_0.vtk");
    vtkFile << "# vtk DataFile Version 3.0\n";
    vtkFile << "Particle Simulation\n";
    vtkFile << "ASCII\n";
    vtkFile << "DATASET POLYDATA\n";
    vtkFile << "POINTS " << numParticles + 8 << " float\n";

    int blockSize = 256;
    int numBlocks = (numParticles + blockSize - 1) / blockSize;

    float TotalTimeforComputation = 0.0f;

    for (int step = 0; step <= num_steps; ++step) 
    {
        hipEvent_t start, stop;
        float elapsedTime;
        hipEventCreate(&start);
        hipEventRecord(start,0);
        
        // 1. Write current positions of all particles to VTK file
        vtkFile << std::fixed << std::setprecision(6);
        for (int i = 0; i < numParticles; ++i) 
        {
            vtkFile << d_positions[i * 3 + 0] << " " << d_positions[i * 3 + 1] << " " << d_positions[i * 3 + 2] << "\n";
        }

        // Write box vertices
        writeBoxVertices(vtkFile, domain);

        
        // 2. Calculate forces
        if (step == 0) 
        {
        calculateForces<<<numBlocks, blockSize>>>(numParticles, d_positions, d_forces, sigma, epsilon, d_domain,cutoff);
        hipDeviceSynchronize();
        }

        // 3. First integration step
        updateParticles<<<numBlocks, blockSize>>>(numParticles, d_positions, d_velocities, d_forces, d_masses, dt, d_domain);
        hipDeviceSynchronize();

        // 4. Calculate new forces
        calculateForces<<<numBlocks, blockSize>>>(numParticles, d_positions, d_forces, sigma, epsilon, d_domain,cutoff);
        hipDeviceSynchronize();

        // 5. Finalize velocities
        finalizeVelocities<<<numBlocks, blockSize>>>(numParticles, d_velocities, d_forces, d_masses, dt);
        hipDeviceSynchronize();

        // 6. Calculate total energy
        hipMemset(d_totalEnergy, 0, sizeof(double));
        calculateTotalEnergy<<<numBlocks, blockSize>>>(numParticles, d_positions, d_velocities, d_masses, sigma, epsilon, d_totalEnergy, d_domain,cutoff);
        hipDeviceSynchronize();

        // Write box edges
        vtkFile << "LINES 12 36\n";
        writeBoxEdges(vtkFile);

        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start,stop);
        TotalTimeforComputation +=elapsedTime; 

        double totalEnergy;
        hipMemcpy(&totalEnergy, d_totalEnergy, sizeof(double), hipMemcpyDeviceToHost);

        if (step % 50 == 0) {
        // Write total energy to file
        std::ofstream energyFile("energy.txt", std::ofstream::app);
        energyFile << "Step: " << step << " Total Energy: " << totalEnergy << "\n";
        energyFile.close();

        // Close and open the VTK file for the next step
        vtkFile.close();
        vtkFile.open("particles_" + std::to_string(step + 1) + ".vtk");
        vtkFile << "# vtk DataFile Version 3.0\n";
        vtkFile << "Particle Simulation\n";
        vtkFile << "ASCII\n";
        vtkFile << "DATASET POLYDATA\n";
        vtkFile << "POINTS " << numParticles + 8 << " float\n";

        std::ofstream outFile("time.txt", std::ofstream::app); 
        outFile << "Time: " << elapsedTime << "ms" << std::endl;
        outFile.close();
        }
    }

    printf("Total time for computaion is : %fms \n", TotalTimeforComputation);

    // Close final VTK file
    vtkFile.close();

    // Free unified memory
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_forces);
    hipFree(d_masses);
    hipFree(d_domain);
    hipFree(d_totalEnergy);

    return 0;
}
